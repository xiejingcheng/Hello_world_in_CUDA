#include "hip/hip_runtime.h"
#include <iostream>
#include "../book.h"
#include <hip/hip_runtime.h>
#define N (33 * 1024)

__global__ void add(int*a ,int *b, int *c){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(){
    int a[N], b[N], c[N];
    int *dev_a ,*dev_b ,*dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a ,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b ,N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c ,N * sizeof(int)));

    HANDLE_ERROR(hipMemset(dev_a, 0, N * sizeof(int)));
    HANDLE_ERROR(hipMemset(dev_b, 0, N * sizeof(int)));
    HANDLE_ERROR(hipMemset(dev_c, 0, N * sizeof(int)));

    for(int i=0 ;i<N;i++){
        a[i] = i;
        b[i] = i*i;
    }

    //注意这里的 dev_a a 方向不要弄反了
    HANDLE_ERROR(hipMemcpy(dev_a,
                            a,
                            N * sizeof(int),
                            hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(dev_b,
                            b,
                            N * sizeof(int),
                            hipMemcpyHostToDevice));
    
    add<<<128,128>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c,
                            dev_c,
                            N * sizeof(int),
                            hipMemcpyDeviceToHost));

    bool success = true;
    for (int i = 0; i<N; i++){
        if((a[i]+b[i])!=c[i]){
            success = false;
        }
    }
    if(success)
    printf("success");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}