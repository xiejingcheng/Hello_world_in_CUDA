#include "hip/hip_runtime.h"
#include <iostream>
#include "../book.h"
#define N (33 * 1024)
#define imin(a,b) (a<b?a:b)

const int threadPerBlock = 128;

const int blockPerGrid = imin(32, (N + threadPerBlock-1)/threadPerBlock);

__global__ void dot(float*a ,float *b, float *c){
    __shared__ float cache[threadPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIdx = threadIdx.x;
    float temp = 0.0;
    while (tid < N){
        temp += a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIdx] = temp;

    __syncthreads();

    int i = blockDim.x/2;
    while(i!=0){
        if(cacheIdx<i){
            cache[cacheIdx] += cache[cacheIdx + i];
        }
        __syncthreads();
        i = i/2;
    }

    if(cacheIdx==0){
        c[blockIdx.x] = cache[0];
    }

}

int main(){
    float count = 0.0;
    float a[N], b[N], c[blockPerGrid];
    float *dev_a ,*dev_b ,*dev_c;
    
    HANDLE_ERROR(hipMalloc((void**)&dev_a ,N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b ,N * sizeof(float)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c ,blockPerGrid * sizeof(float)));

    HANDLE_ERROR(hipMemset(dev_a, 0, N * sizeof(float)));
    HANDLE_ERROR(hipMemset(dev_b, 0, N * sizeof(float)));
    HANDLE_ERROR(hipMemset(dev_c, 0, blockPerGrid * sizeof(float)));

    for(int i=0 ;i<N;i++){
        a[i] = i;
        b[i] = i*i;
    }

    HANDLE_ERROR(hipMemcpy(dev_a,
        a,
        N * sizeof(float),
        hipMemcpyHostToDevice));

    HANDLE_ERROR(hipMemcpy(dev_b,
        b,
        N * sizeof(float),
        hipMemcpyHostToDevice));

    dot<<<blockPerGrid, threadPerBlock>>>(dev_a, dev_b, dev_c);

    HANDLE_ERROR(hipMemcpy(c,
        dev_c,
        blockPerGrid * sizeof(float),
        hipMemcpyDeviceToHost));
    
    for(int i =0 ;i < blockPerGrid; i++){
        count += c[i];
    }

    printf("%f",count);


    return 0;
}