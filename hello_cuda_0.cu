#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"
__global__ void hello()
{
    printf("Hello CUDA\n");
}


int main() {
    hello<<<1,1>>>();
    hipDeviceSynchronize();
    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));

    printf("hipGetDeviceCount:%d",count);
    return 0;


}
