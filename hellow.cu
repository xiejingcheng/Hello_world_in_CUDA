#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
    printf("Hello CUDA\n");
}

int main() {
    hello<<<1,1000 >>>();
    hipDeviceSynchronize();
    return 0;
}
